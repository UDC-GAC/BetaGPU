#include "hip/hip_runtime.h"
#include "../src_ref/BetaDistGsl.hpp"

#include <chrono>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <vector>

#include <omp.h>

using std::vector;
using std::cerr;
using std::endl;

using profile_clock_t = std::chrono::high_resolution_clock;
using profile_duration_t = std::chrono::duration<double>;





struct CommandLineOptions {

  enum class ExecutionMode {
    SEQ,
    OMP,
    CUDA
  };

  enum class FunctionName {
    BETAPDF,
    BETACDF
  };

  int num_elements;
  int num_iterations;
  ExecutionMode exec_mode;
  FunctionName function_name;
};

__global__ void betapdf_kernel_self(double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = pow(x[idx], alpha - 1) * pow(1 - x[idx], beta - 1) * exp(lgamma(alpha + beta) - lgamma(alpha) - lgamma(beta));
    }
}

// CUDA kernel launch to compute the beta distribution
std::vector<double> betapdf_cuda_self(std::vector<double> x, double alpha, double beta){
    // Allocate memory on the device
    double *d_x, *d_y;
    hipMalloc(&d_x, x.size() * sizeof(double));
    hipMalloc(&d_y, x.size() * sizeof(double));

    // Copy the data to the device
    hipMemcpy(d_x, x.data(), x.size() * sizeof(double), hipMemcpyHostToDevice);

    // Launch the kernel
    int block_size = 256;
    int n_blocks = x.size() / block_size + (x.size() % block_size == 0 ? 0 : 1);
    betapdf_kernel_self<<<n_blocks, block_size>>>(d_x, d_y, alpha, beta, x.size());

    // Copy the result back to the host
    std::vector<double> y(x.size());
    hipMemcpy(y.data(), d_y, x.size() * sizeof(double), hipMemcpyDeviceToHost);

    // Free the memory on the device
    hipFree(d_x);
    hipFree(d_y);

    return y;
}

static std::string get_help_message(std::string prog_name) {
  return "Usage: " + prog_name + R"( [num_elements] [num_iterations] [exec_mode] [function_name]
  num_elements: Number of elements in the input vector
  num_iterations: Number of iterations to run the test
  exec_mode: Execution mode (seq, omp, cuda)
  function_name: Name of the function to test (betapdf, betacdf)
)";
}

int parse_positive_int(char* str) {
  int value = std::stoi(str);
  if (value <= 0) {
    throw std::invalid_argument("Value must be a positive integer");
  }
  return value;
}

CommandLineOptions::ExecutionMode parse_exec_mode(char* str) {
  std::string mode(str);
  if (mode == "seq")
    return CommandLineOptions::ExecutionMode::SEQ;
  if (mode == "omp")
    return CommandLineOptions::ExecutionMode::OMP;
  if (mode == "cuda")
    return CommandLineOptions::ExecutionMode::CUDA;
  
  throw std::invalid_argument("Invalid execution mode" + mode + "\n\t Valid execution modes are: seq, omp, cuda");
}

CommandLineOptions::FunctionName parse_function_name(char* str) {
  std::string mode(str);
  if (mode == "betapdf")
    return CommandLineOptions::FunctionName::BETAPDF;
  if (mode == "betacdf")
    return CommandLineOptions::FunctionName::BETACDF;
  
  throw std::invalid_argument("Invalid function name" + mode + "\n\t Valid function names are: betapdf, betacdf");
}
  
CommandLineOptions parse_command_line(int argc, char *argv[]) {
  CommandLineOptions options;
  if (argc != 5) {
    std::cerr << get_help_message(argv[0]) << std::endl;
    exit(EXIT_SUCCESS);
  }

  options.num_elements = parse_positive_int(argv[1]);
  options.num_iterations = parse_positive_int(argv[2]);
  options.exec_mode = parse_exec_mode(argv[3]);
  options.function_name = parse_function_name(argv[4]);

  return options;
}

std::string mode_to_text(CommandLineOptions::ExecutionMode mode) {
  switch (mode) {
  case CommandLineOptions::ExecutionMode::SEQ:
    return "Sequential";
  case CommandLineOptions::ExecutionMode::OMP:
    return "OpenMP";
  case CommandLineOptions::ExecutionMode::CUDA:
    return "CUDA";
  }
  return "Unknown";
}

std::string function_to_test(CommandLineOptions::FunctionName function) {
  switch (function) {
  case CommandLineOptions::FunctionName::BETAPDF:
    return "Beta PDF";
  case CommandLineOptions::FunctionName::BETACDF:
    return "Beta CDF";
  }
  return "Unknown";
}

void print_execution_parameters(const CommandLineOptions& options) {
  cerr << "+------------------------------------+" << endl;
  cerr << "|        Execution Parameters        |" << endl;
  cerr << "+------------------------------------+" << endl;
  cerr << "\tNumber of elements: " << options.num_elements << endl;
  cerr << "\tNumber of iterations: " << options.num_iterations << endl;
  cerr << "\tExecution mode: " << mode_to_text(options.exec_mode) << endl;
  cerr << "\tFunction name: " << function_to_test(options.function_name) << endl;
}

void execute_test(const CommandLineOptions& options, vector<double>& x, double alpha, double beta){
  vector<double> y(x.size());
  switch (options.exec_mode) {
  case CommandLineOptions::ExecutionMode::SEQ:
    switch (options.function_name) {
    case CommandLineOptions::FunctionName::BETAPDF:
      for (size_t i = 0; i < x.size(); i++) {
        y.at(i) = betapdf(x.at(i), alpha, beta);
      }
      break;
    case CommandLineOptions::FunctionName::BETACDF:
      for (size_t i = 0; i < x.size(); i++) {
        y.at(i) = betacdf(x.at(i), alpha, beta);
      }
      break;
    }
    break;
  case CommandLineOptions::ExecutionMode::OMP:
    switch (options.function_name) {
    case CommandLineOptions::FunctionName::BETAPDF:
      #pragma omp parallel for schedule(static, 64)
      for (size_t i = 0; i < x.size(); i++) {
        y.at(i) = betacdf(x.at(i), alpha, beta);
      }
      break;
    case CommandLineOptions::FunctionName::BETACDF:
      #pragma omp parallel for schedule(static, 64)
      for (size_t i = 0; i < x.size(); i++) {
        y.at(i) = betacdf(x.at(i), alpha, beta);
      }
      break;
    }
    break;
  case CommandLineOptions::ExecutionMode::CUDA:
    switch (options.function_name) {
    case CommandLineOptions::FunctionName::BETAPDF:
      cerr << "hello" << endl;
      y = betapdf_cuda_self(x, alpha, beta);
      break;
    case CommandLineOptions::FunctionName::BETACDF:
      y = betapdf_cuda_self(x, alpha, beta);
      break;
    }
    break;
  }
}

int 
main (int argc, char *argv[]) {

  CommandLineOptions options = parse_command_line(argc, argv);
  print_execution_parameters(options);
  
  vector<double> x(options.num_elements);
  

  for (int i = 0; i < options.num_elements; i++) {
    x[i] = rand() / (double)RAND_MAX;
  }

  auto full_start = profile_clock_t::now();
  for (int i = 1; i <= options.num_iterations; i++) {
    double alpha = 0.1 * i;
    double beta = 0.1 * i;
    
    auto start = profile_clock_t::now();
    execute_test(options, x, alpha, beta);
    auto end = profile_clock_t::now();

    cerr << "Itr[" << i << "]\t\tTime = \t\t" << profile_duration_t(end - start).count() << endl;
  }
  auto full_end = profile_clock_t::now();
  cerr << "Total time = " << profile_duration_t(full_end - full_start).count() << endl;
  
}
