#include "hip/hip_runtime.h"
#include "beta_cuda/BetaDistCuda.hpp"

#include <hip/hip_fp16.h>
#include <gsl/gsl_sf_gamma.h>
#include <stdexcept>

#include <omp.h>
  
#ifdef DEBUG

#include <chrono>
#include <iostream>

using std::cerr;
using std::endl;

using profile_clock_t = std::chrono::high_resolution_clock;
using profile_duration_t = std::chrono::duration<double>;

#endif

// Define a type for the function that launches the kernel
typedef void (*KernelLauncher)(const double*, double*, double, double, int, int, hipStream_t stream);
typedef void (*KernelLauncherFloat)(const float*, float*, float, float, int, int, hipStream_t stream);

const hipStream_t CUDA_DEFAULT_STREAM = (hipStream_t) 0;
constexpr int DEFAULT_BLOCK_SIZE = 256;


/* --------------- Beta PDF Kernels --------------- */


__global__ void betapdf_kernel(const double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = pow(x[idx], alpha - 1) * pow(1 - x[idx], beta - 1) * exp(lgamma(alpha + beta) - lgamma(alpha) - lgamma(beta));
    }
}

__global__ void betapdf_kernel_f(const float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}

__global__ void betapdf_kernel_h(const float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}


/* --------------- Beta CDF Kernels --------------- */


// Look https://github.com/ampl/gsl/blob/master/cdf/beta_inc.c#L26
__device__ __host__ double cuda_beta_cont_frac (const double a, const double b, const double x,
                const double epsabs) {
  const unsigned int max_iter = 512;    /* control iterations      */
  const double cutoff = 2.0 * CUDA_DBL_MIN;      /* control the zero cutoff */
  unsigned int iter_count = 0;
  double cf;

  /* standard initialization for continued fraction */
  double num_term = 1.0;
  double den_term = 1.0 - (a + b) * x / (a + 1.0);

  if (fabs (den_term) < cutoff)
    den_term = nan("");

  den_term = 1.0 / den_term;
  cf = den_term;

  while (iter_count < max_iter)
    {
      const int k = iter_count + 1;
      double coeff = k * (b - k) * x / (((a - 1.0) + 2 * k) * (a + 2 * k));
      double delta_frac;

      /* first step */
      den_term = 1.0 + coeff * den_term;
      num_term = 1.0 + coeff / num_term;

      if (fabs (den_term) < cutoff)
        den_term = nan("");

      if (fabs (num_term) < cutoff)
        num_term = nan("");

      den_term = 1.0 / den_term;

      delta_frac = den_term * num_term;
      cf *= delta_frac;

      coeff = -(a + k) * (a + b + k) * x / ((a + 2 * k) * (a + 2 * k + 1.0));

      /* second step */
      den_term = 1.0 + coeff * den_term;
      num_term = 1.0 + coeff / num_term;

      if (fabs (den_term) < cutoff)
        den_term = nan("");

      if (fabs (num_term) < cutoff)
        num_term = nan("");

      den_term = 1.0 / den_term;

      delta_frac = den_term * num_term;
      cf *= delta_frac;

      if (fabs (delta_frac - 1.0) < 2.0 * CUDA_DBL_EPSILON)
        break;

      if (cf * fabs (delta_frac - 1.0) < epsabs)
        break;

      ++iter_count;
    }

  if (iter_count >= max_iter)
    return nan("");

  return cf;
}

__global__ void betacdf_dirCF_kernel(double *x, double *y, double alpha, double beta, double ln_beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double a = alpha;
    double b = beta;
    if (idx < size){
        double my_x = x[idx];

        double ln_pre = -ln_beta + a * log(my_x) + b * log1p(-my_x);
        double prefactor = exp(ln_pre);

        double epsabs = 0.;
        double cf = cuda_beta_cont_frac(a, b, my_x, epsabs);

        y[idx] = prefactor * cf / a;
        
    }
}

__global__ void betacdf_hypergeoCF_kernel(double *x, double *y, double alpha, double beta, double ln_beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double a = alpha;
    double b = beta;
    if (idx < size){
        double my_x = x[idx];

        double ln_pre = -ln_beta + a * log(my_x) + b * log1p(-my_x);
        double prefactor = exp(ln_pre);

        double epsabs = 1. / (prefactor / b) * CUDA_DBL_EPSILON;
        double cf = cuda_beta_cont_frac(b, a, 1. - my_x, epsabs);

        double term = prefactor * cf / b;

        y[idx] = 1. - term;
    }
}

__global__ void betacdf_CF_kernel(const double *x, double *y, double alpha, double beta, double ln_beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double a = alpha;
    double b = beta;
    double limit = (alpha + 1.0) / (alpha + beta + 2.0);
    if (idx < size){
        double my_x = x[idx];

        double ln_pre = -ln_beta + a * log(my_x) + b * log1p(-my_x);
        double prefactor = exp(ln_pre);

        double epsabs = my_x < limit ? 0. : 1. / (prefactor / b) * CUDA_DBL_EPSILON; // Now every value can be one of two cases
        double cf_a = my_x < limit ? a : b;
        double cf_b = my_x < limit ? b : a;
        double cf_x = my_x < limit ? my_x : 1. - my_x;
        double cf = cuda_beta_cont_frac(cf_a, cf_b, cf_x, epsabs);

        double term = prefactor * cf / cf_a;

        double my_y = my_x < limit ? term : 1. - term;
        y[idx] =  my_y; // Now every value can be one of two cases
    }
}

// https://github.com/ampl/gsl/blob/master/specfunc/gamma_inc.c#L500
__global__ void betacdf_la_sb_kernel(double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double a = alpha;
    double b = beta;
    if (idx < size){
        double my_x = x[idx];
        double N = a + (b - 1.) / 2.;
        my_x = -N * log1p(-my_x);

        y[idx] = nan("");
        return;
    }
}

// https://github.com/ampl/gsl/blob/master/specfunc/gamma_inc.c#L581
__global__ void betacdf_sa_lb_kernel_f(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float a = alpha;
    float b = beta;
    if (idx < size){
        float my_x = x[idx];
        float N = b + (a - 1.) / 2.;
        my_x = -N * log1pf(-my_x);

        y[idx] = nanf("");
        return;
    }
}

__global__ void betacdf_prefix_only_kernel(const double *x, double *y, double alpha, double beta, double ln_beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double a = alpha;
    double b = beta;
    if (idx < size){
        double my_x = x[idx];

        double ln_pre = -ln_beta + a * log(my_x) + b * log1p(-my_x);
        double prefactor = exp(ln_pre);

        y[idx] = prefactor;
    }
}


/* --------------- Kernel launchers --------------- */


inline void launch_betapdf_kernel(const double *d_x, double *d_y, double alpha, double beta, int size, int block_size, hipStream_t stream=CUDA_DEFAULT_STREAM) {
    int n_blocks = size / block_size + (size % block_size == 0 ? 0 : 1);
    betapdf_kernel<<<n_blocks, block_size,0,stream>>>(d_x, d_y, alpha, beta, size);

}

inline void launch_betapdf_kernel_f(const float *d_x, float *d_y, float alpha, float beta, int size, int block_size, hipStream_t stream=CUDA_DEFAULT_STREAM) {
    int n_blocks = size / block_size + (size % block_size == 0 ? 0 : 1);
    betapdf_kernel_f<<<n_blocks, block_size,0,stream>>>(d_x, d_y, alpha, beta, size);
}

inline void launch_betacdf_prefactor_only_kernel(const double *d_x, double *d_y, double alpha, double beta, int size, int block_size, hipStream_t stream=CUDA_DEFAULT_STREAM) {
    int n_blocks = size / block_size + (size % block_size == 0 ? 0 : 1);
    double ln_beta = gsl_sf_lnbeta(alpha, beta);
    betacdf_prefix_only_kernel<<<n_blocks, block_size,0,stream>>>(d_x, d_y, alpha, beta, ln_beta, size);
}

inline void launch_betacdf_withCF_kernel(const double *d_x, double *d_y, double alpha, double beta, int size, int block_size, hipStream_t stream=CUDA_DEFAULT_STREAM) {
    int n_blocks = size / block_size + (size % block_size == 0 ? 0 : 1);
    double ln_beta = gsl_sf_lnbeta(alpha, beta);
    betacdf_CF_kernel<<<n_blocks, block_size,0,stream>>>(d_x, d_y, alpha, beta, ln_beta, size);
}


/* --------------- Auxiliar encapsulation functions --------------- */

size_t get_free_GPU_memory(){
  size_t free_bytes, total_bytes;

  hipMemGetInfo( &free_bytes, &total_bytes);

  #ifdef DEBUG
    size_t used_bytes = total_bytes - free_bytes;
    std::cerr << "GPU memory usage: " << (used_bytes>>20) << " bytes used, " << (free_bytes>>20) << " MegaBytes free, " << (total_bytes>>20) << " MegaBytes total." << std::endl;
  #endif

  return free_bytes;
}

template <typename T, typename K>
void beta_array_cuda(const T *x, T *y, const T alpha, const T beta, unsigned long size, K kernel_launcher){

  #ifdef DEBUG
    hipEvent_t t1, t2, t3, t4;
    float elapsedMemcpyCG, elapsedKernel, elapsedMemcpyGC, elapsedTotal;
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventCreate(&t3);
    hipEventCreate(&t4);

    hipEventRecord(t1, 0);
  #endif

  // Allocate memory on the device
  T *d_x, *d_y;

  hipMalloc(&d_x, size * sizeof(T));
  hipMalloc(&d_y, size * sizeof(T));

  // Copy the data to the device
  hipMemcpy(d_x, x, size * sizeof(T), hipMemcpyHostToDevice);

  #ifdef DEBUG
    hipEventRecord(t2, 0);
    hipEventSynchronize(t2);
    hipEventElapsedTime(&elapsedMemcpyCG, t1, t2);
  #endif

  // Launch the kernel
  int block_size = DEFAULT_BLOCK_SIZE;
  kernel_launcher(d_x, d_y, alpha, beta, size, block_size, CUDA_DEFAULT_STREAM);

  #ifdef DEBUG
    hipEventRecord(t3, 0);
    hipEventSynchronize(t3);
    hipEventElapsedTime(&elapsedKernel, t2, t3);
  #endif

  // Copy the result back to the host
  hipMemcpy(y, d_y, size * sizeof(T), hipMemcpyDeviceToHost);

  // Free the memory on the device
  hipFree(d_x);
  hipFree(d_y);

  #ifdef DEBUG
    hipEventRecord(t4, 0);
    hipEventSynchronize(t4);
    hipEventElapsedTime(&elapsedMemcpyGC, t3, t4);
    hipEventElapsedTime(&elapsedTotal, t1, t4);

    cerr << "Full function time(events) = " << elapsedTotal / 1000 << endl;
    cerr << "\tKernel execution time = " << elapsedKernel / 1000 << endl;
    cerr << "\tMemory transfer time = " << elapsedMemcpyCG / 1000 << " + " << elapsedMemcpyGC / 1000 << endl;
  #endif

  return;
}

template <typename T, typename K>
void beta_array_cuda_streams(const T *x, T *y, const T alpha, const T beta, unsigned long size, K kernel_launcher, const unsigned int num_streams=2, const unsigned int chunks_per_stream=2){

  // Allocate memory on the device
  T *d_x[num_streams], *d_y[num_streams];

  unsigned long chunk_size = size / (chunks_per_stream * num_streams);
  unsigned long remainer_elements = size % (chunks_per_stream * num_streams);

  // Create streams
  hipStream_t streams[num_streams];
  for (unsigned int i = 0; i < num_streams; i++){
    hipStreamCreate(&streams[i]);

    unsigned long max_chunk_size = remainer_elements ? chunk_size + 1 : chunk_size;

    // Allocate memory on the device
    hipMalloc(&d_x[i], (max_chunk_size) * sizeof(T));
    hipMalloc(&d_y[i], (max_chunk_size) * sizeof(T));
  }

  // Work 
  unsigned long start_offset = 0;
  for (int block_idx = 0; block_idx < chunks_per_stream * num_streams; block_idx++){
    int stream_idx = block_idx % num_streams;
    unsigned long current_chunk_size = block_idx < remainer_elements ? chunk_size + 1 : chunk_size;

    // Copy the data to the device
    hipMemcpyAsync(d_x[stream_idx], x + start_offset, current_chunk_size * sizeof(T), hipMemcpyHostToDevice, streams[stream_idx]);

    kernel_launcher(d_x[stream_idx], d_y[stream_idx], alpha, beta, current_chunk_size, 256, streams[stream_idx]);

    hipMemcpyAsync(y + start_offset, d_y[stream_idx], current_chunk_size * sizeof(T), hipMemcpyDeviceToHost, streams[stream_idx]);

    start_offset += current_chunk_size;
  }

  // Destroy streams
  for (unsigned int i = 0; i < num_streams; i++){
    hipStreamDestroy(streams[i]);

    // Free the memory on the device
    hipFree(d_x[i]);
    hipFree(d_y[i]);
  }

  return;

}

template <typename T, typename K>
void beta_array_cuda_wrapper (const T *x, T *y, const T alpha, const T beta, unsigned long size, K kernel_launcher){
  size_t free_bytes = get_free_GPU_memory();
  size_t needed_bytes = ((2 * size ) + 2)* sizeof(double); // 2 arrays of size +
                                                           // 1 extra element per stream 
                                                           // (for odd distributions to fill the remainer element, 2 streams fixed)

  #ifdef DEBUG
    std::cerr << "! Executing GPU function." << std::endl;
    std::cerr << "+-- Needed memory: " << (needed_bytes>>20) << " MegaBytes." << std::endl;
    std::cerr << "+-- Free memory: " << (free_bytes>>20) << " MegaBytes." << std::endl;
  #endif
  if (needed_bytes < free_bytes){
    // If i have enough memory, use the normal function
    #ifdef DEBUG
      std::cerr << "+- Using normal function." << std::endl;
    #endif
    beta_array_cuda<T, K>(x, y, alpha, beta, size, kernel_launcher);
  } else {
    // If i don't have enough memory, use streams
    unsigned int chunks_per_stream = (needed_bytes / free_bytes) + 1;
    #ifdef DEBUG
      std::cerr << "+- Using streams with " << chunks_per_stream << " chunks per stream." << std::endl;
    #endif
    beta_array_cuda_streams<T, K>(x, y, alpha, beta, size, kernel_launcher, 2, chunks_per_stream);
  }

  return;
}

/* --------------- Export fuctions --------------- */


// CUDA kernel launch to compute the beta distribution
void betapdf_cuda(const double *x, double *y, const double alpha, const double beta, unsigned long size, Memory_Type memory_type){

  if (memory_type == Memory_Type::HOST){
    beta_array_cuda_wrapper<double, KernelLauncher>(x, y, alpha, beta, size, launch_betapdf_kernel);
  } 

  // If the memory type is DEVICE, we can use KernelLauncher function directly
  if (memory_type == Memory_Type::DEVICE){
    launch_betapdf_kernel(x, y, alpha, beta, size, DEFAULT_BLOCK_SIZE);
  }
  

  return;
}

// CUDA kernel launch to compute the beta distribution
void betapdf_cuda(const float *x, float *y, const float alpha, const float beta, unsigned long size, Memory_Type memory_type){

  if (memory_type == Memory_Type::HOST){
    beta_array_cuda_wrapper<float, KernelLauncherFloat>(x, y, alpha, beta, size, launch_betapdf_kernel_f);
  }

  // If the memory type is DEVICE, we can use KernelLauncher function directly
  if (memory_type == Memory_Type::DEVICE){
    launch_betapdf_kernel_f(x, y, alpha, beta, size, DEFAULT_BLOCK_SIZE);
  }

  return;
}

// CUDA kernel launch to compute the beta distribution
void betacdf_cuda(const double *x, double *y, const double alpha, const double beta, unsigned long size, Memory_Type memory_type){

  if (memory_type == Memory_Type::HOST){
    beta_array_cuda_wrapper<double, KernelLauncher>(x, y, alpha, beta, size, launch_betacdf_withCF_kernel);
  }

  // If the memory type is DEVICE, we can use KernelLauncher function directly
  if (memory_type == Memory_Type::DEVICE){
    launch_betacdf_withCF_kernel(x, y, alpha, beta, size, DEFAULT_BLOCK_SIZE);
  }

  return;
}

void betacdf_cuda_GPU_CPU(const double *x, double *y, const double alpha, const double beta, unsigned long size, Memory_Type memory_type){

  if (memory_type == Memory_Type::HOST) {
    beta_array_cuda_wrapper<double, KernelLauncher>(x, y, alpha, beta, size, launch_betacdf_prefactor_only_kernel);
  }

  // If the memory type is DEVICE, we can use KernelLauncher function directly
  if (memory_type == Memory_Type::DEVICE) {
    throw std::runtime_error("This function is not implemented for DEVICE memory type.");
  }

#pragma omp parallel for schedule(static, 64)
  for (unsigned long i = 0; i < size; i++) {
    if (x[i] < (alpha + 1.0) / (alpha + beta + 2.0)) {
      /* Apply continued fraction directly. */
      double epsabs = 0.;

      double cf = cuda_beta_cont_frac(alpha, beta, x[i], epsabs);

      y[i] = y[i] * cf / alpha;
    } else {
      /* Apply continued fraction after hypergeometric transformation. */
      double epsabs =
          fabs(1. / (y[i] / beta)) * CUDA_DBL_EPSILON;
      double cf = cuda_beta_cont_frac(beta, alpha, 1.0 - x[i], epsabs);
      double term = y[i] * cf / beta;

      y[i] = 1 - term;
    }
  }

  return;
}
