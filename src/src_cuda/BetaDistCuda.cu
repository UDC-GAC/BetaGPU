#include "hip/hip_runtime.h"
#include "BetaDistCuda.hpp"

#include <hip/hip_fp16.h>


__global__ void betapdf_kernel(double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = pow(x[idx], alpha - 1) * pow(1 - x[idx], beta - 1) * exp(lgamma(alpha + beta) - lgamma(alpha) - lgamma(beta));
    }
}

__global__ void betapdf_kernel_f(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}

__global__ void betapdf_kernel_h(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}

// CUDA kernel launch to compute the beta distribution
std::vector<double> betapdf_cuda(std::vector<double> x, double alpha, double beta, GPU_Type precision){
    // Allocate memory on the device
    double *d_x, *d_y;
    float *d_x_f, *d_y_f, alpha_f, beta_f;
    if (precision == GPU_Type::DOUBLE){
        hipMalloc(&d_x, x.size() * sizeof(double));
        hipMalloc(&d_y, x.size() * sizeof(double));
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        alpha_f = (float)alpha;
        beta_f = (float)beta;
        hipMalloc(&d_x_f, x.size() * sizeof(float));
        hipMalloc(&d_y_f, x.size() * sizeof(float));
    }

    // Copy the data to the device
    if (precision == GPU_Type::DOUBLE){
        hipMemcpy(d_x, x.data(), x.size() * sizeof(double), hipMemcpyHostToDevice);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> x_f(x.begin(), x.end());
        hipMemcpy(d_x_f, x_f.data(), x_f.size() * sizeof(float), hipMemcpyHostToDevice);
    }

    // Launch the kernel
    int block_size = 256;
    int n_blocks = x.size() / block_size + (x.size() % block_size == 0 ? 0 : 1);
    if (precision == GPU_Type::DOUBLE)
        betapdf_kernel<<<n_blocks, block_size>>>(d_x, d_y, alpha, beta, x.size());
    if (precision == GPU_Type::FLOAT)
        betapdf_kernel_f<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha, beta, x.size());
    if (precision == GPU_Type::HALF)
        betapdf_kernel_h<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha_f, beta_f, x.size());

    // Copy the result back to the host
    std::vector<double> y(x.size());
    if (precision == GPU_Type::DOUBLE)
        hipMemcpy(y.data(), d_y, x.size() * sizeof(double), hipMemcpyDeviceToHost);
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> y_f(x.size());
        hipMemcpy(y_f.data(), d_y_f, x.size() * sizeof(float), hipMemcpyDeviceToHost);
        y = std::vector<double>(y_f.begin(), y_f.end());
    }

    // Free the memory on the device
    if (precision == GPU_Type::DOUBLE){
        hipFree(d_x);
        hipFree(d_y);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        hipFree(d_x_f);
        hipFree(d_y_f);
    }

    return y;
}

std::vector<double> betapdf_cuda_f(std::vector<double> x, double alpha, double beta){
    float alpha_f = (float)alpha;
    float beta_f = (float)beta;
    std::vector<float> x_f(x.begin(), x.end());

    // Allocate memory on the device
    float *d_x, *d_y;
    hipMalloc(&d_x, x_f.size() * sizeof(float));
    hipMalloc(&d_y, x_f.size() * sizeof(float));

    // Copy the data to the device
    hipMemcpy(d_x, x_f.data(), x_f.size() * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    int block_size = 256;
    int n_blocks = x_f.size() / block_size + (x_f.size() % block_size == 0 ? 0 : 1);
    betapdf_kernel_f<<<n_blocks, block_size>>>(d_x, d_y, alpha_f, beta_f, x_f.size());

    // Copy the result back to the host
    std::vector<float> y_f(x_f.size());
    hipMemcpy(y_f.data(), d_y, x_f.size() * sizeof(float), hipMemcpyDeviceToHost);

    // Free the memory on the device
    hipFree(d_x);
    hipFree(d_y);

    return std::vector<double>(y_f.begin(), y_f.end());
}


std::vector<double> betacdf_cuda(std::vector<double> x, double alpha, double beta){
    return std::vector<double>();
}
