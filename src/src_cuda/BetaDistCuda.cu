#include "hip/hip_runtime.h"
#include "BetaDistCuda.hpp"

#include <hip/hip_fp16.h>

#ifdef DEBUG

#include <chrono>
#include <iostream>

using std::cerr;
using std::endl;

using profile_clock_t = std::chrono::high_resolution_clock;
using profile_duration_t = std::chrono::duration<double>;

#endif

__global__ void betapdf_kernel(double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = pow(x[idx], alpha - 1) * pow(1 - x[idx], beta - 1) * exp(lgamma(alpha + beta) - lgamma(alpha) - lgamma(beta));
    }
}

__global__ void betapdf_kernel_f(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}

__global__ void betapdf_kernel_h(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}

// CUDA kernel launch to compute the beta distribution
std::vector<double> betapdf_cuda(std::vector<double> x, double alpha, double beta, GPU_Type precision){
    // Allocate memory on the device
    double *d_x, *d_y;
    float *d_x_f, *d_y_f, alpha_f, beta_f;
    if (precision == GPU_Type::DOUBLE){
        hipMalloc(&d_x, x.size() * sizeof(double));
        hipMalloc(&d_y, x.size() * sizeof(double));
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        alpha_f = (float)alpha;
        beta_f = (float)beta;
        hipMalloc(&d_x_f, x.size() * sizeof(float));
        hipMalloc(&d_y_f, x.size() * sizeof(float));
    }

    // Copy the data to the device
    if (precision == GPU_Type::DOUBLE){
        hipMemcpy(d_x, x.data(), x.size() * sizeof(double), hipMemcpyHostToDevice);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> x_f(x.begin(), x.end());
        hipMemcpy(d_x_f, x_f.data(), x_f.size() * sizeof(float), hipMemcpyHostToDevice);
    }

    // Launch the kernel
    int block_size = 256;
    int n_blocks = x.size() / block_size + (x.size() % block_size == 0 ? 0 : 1);
    if (precision == GPU_Type::DOUBLE)
        betapdf_kernel<<<n_blocks, block_size>>>(d_x, d_y, alpha, beta, x.size());
    if (precision == GPU_Type::FLOAT)
        betapdf_kernel_f<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha, beta, x.size());
    if (precision == GPU_Type::HALF)
        betapdf_kernel_h<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha_f, beta_f, x.size());

    // Copy the result back to the host
    std::vector<double> y(x.size());
    if (precision == GPU_Type::DOUBLE)
        hipMemcpy(y.data(), d_y, x.size() * sizeof(double), hipMemcpyDeviceToHost);
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> y_f(x.size());
        hipMemcpy(y_f.data(), d_y_f, x.size() * sizeof(float), hipMemcpyDeviceToHost);
        y = std::vector<double>(y_f.begin(), y_f.end());
    }

    // Free the memory on the device
    if (precision == GPU_Type::DOUBLE){
        hipFree(d_x);
        hipFree(d_y);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        hipFree(d_x_f);
        hipFree(d_y_f);
    }

    return y;
}

#ifdef DEBUG
std::vector<double> betapdf_cuda_times(std::vector<double> x, double alpha, double beta, GPU_Type precision){
    hipEvent_t t1, t2, t3, t4;
    float elapsedMemcpyCG, elapsedKernel, elapsedMemcpyGC, elapsedTotal;
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventCreate(&t3);
    hipEventCreate(&t4);

    auto start = profile_clock_t::now();

    hipEventRecord(t1, 0);
    // Allocate memory on the device
    double *d_x, *d_y;
    float *d_x_f, *d_y_f, alpha_f, beta_f;
    if (precision == GPU_Type::DOUBLE){
        hipMalloc(&d_x, x.size() * sizeof(double));
        hipMalloc(&d_y, x.size() * sizeof(double));
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        alpha_f = (float)alpha;
        beta_f = (float)beta;
        hipMalloc(&d_x_f, x.size() * sizeof(float));
        hipMalloc(&d_y_f, x.size() * sizeof(float));
    }

    // Copy the data to the device
    if (precision == GPU_Type::DOUBLE){
        hipMemcpy(d_x, x.data(), x.size() * sizeof(double), hipMemcpyHostToDevice);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> x_f(x.begin(), x.end());
        hipMemcpy(d_x_f, x_f.data(), x_f.size() * sizeof(float), hipMemcpyHostToDevice);
    }

    hipEventRecord(t2, 0);
    hipEventSynchronize(t2);
    hipEventElapsedTime(&elapsedMemcpyCG, t1, t2);

    // Launch the kernel
    int block_size = 256;
    int n_blocks = x.size() / block_size + (x.size() % block_size == 0 ? 0 : 1);
    if (precision == GPU_Type::DOUBLE)
        betapdf_kernel<<<n_blocks, block_size>>>(d_x, d_y, alpha, beta, x.size());
    if (precision == GPU_Type::FLOAT)
        betapdf_kernel_f<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha, beta, x.size());
    if (precision == GPU_Type::HALF)
        betapdf_kernel_h<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha_f, beta_f, x.size());

    hipEventRecord(t3, 0);
    hipEventSynchronize(t3);
    hipEventElapsedTime(&elapsedKernel, t2, t3);

    // Copy the result back to the host
    std::vector<double> y(x.size());
    if (precision == GPU_Type::DOUBLE)
        hipMemcpy(y.data(), d_y, x.size() * sizeof(double), hipMemcpyDeviceToHost);
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> y_f(x.size());
        hipMemcpy(y_f.data(), d_y_f, x.size() * sizeof(float), hipMemcpyDeviceToHost);
        y = std::vector<double>(y_f.begin(), y_f.end());
    }

    // Free the memory on the device
    if (precision == GPU_Type::DOUBLE){
        hipFree(d_x);
        hipFree(d_y);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        hipFree(d_x_f);
        hipFree(d_y_f);
    }

    hipEventRecord(t4, 0);
    hipEventSynchronize(t4);
    hipEventElapsedTime(&elapsedMemcpyGC, t3, t4);
    hipEventElapsedTime(&elapsedTotal, t1, t4);

    auto end = profile_clock_t::now();

    cerr << "Full function time(chrono) = " << profile_duration_t(end - start).count() << endl;
    cerr << "Full function time(events) = " << elapsedTotal / 1000 << endl;
    cerr << " Kernel execution time = " << elapsedKernel / 1000 << endl;
    cerr << " Memory transfer time = " << elapsedMemcpyCG / 1000 << " + " << elapsedMemcpyGC / 1000 << endl;

    return y;
}
#endif

std::vector<double> betacdf_cuda(std::vector<double> x, double alpha, double beta){
    return std::vector<double>();
}
