#include "hip/hip_runtime.h"
#include "BetaDistCuda.hpp"

#include <hip/hip_fp16.h>

#ifdef DEBUG

#include <chrono>
#include <iostream>

using std::cerr;
using std::endl;

using profile_clock_t = std::chrono::high_resolution_clock;
using profile_duration_t = std::chrono::duration<double>;

#endif

__global__ void betapdf_kernel(double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = pow(x[idx], alpha - 1) * pow(1 - x[idx], beta - 1) * exp(lgamma(alpha + beta) - lgamma(alpha) - lgamma(beta));
    }
}

__global__ void betapdf_kernel_f(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}

__global__ void betapdf_kernel_h(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}

// TODO: Implement the beta distribution CDF using the continued fraction
__device__ double cuda_beta_cont_frac(double alpha, double beta, double x, double epsabs){
    const unsigned int max_iter = 512;
    const double cutoff = 2. * CUDA_DBL_MIN;
    double cf;
    double delta_frac;

    double num_term = 1.;
    double denom_term = 1. - (alpha + beta) * x / (alpha + 1.);

    if (fabs(denom_term) < cutoff)
        denom_term = nan("");

    denom_term = 1. / denom_term;
    cf = denom_term;

    for (unsigned int iter = 0; iter < max_iter; iter++){
        
        const unsigned int k = iter + 1;
        double coeff = k * (beta - k) * x / (((alpha - 1.) + 2 * k) * (alpha + 2 * k));
        

        /* first step */
        denom_term = 1. + coeff * denom_term;
        num_term = 1. + coeff / num_term;

        if (fabs(denom_term) < cutoff)
            denom_term = nan("");

        if (fabs(num_term) < cutoff)
            num_term = nan("");

        denom_term = 1. / denom_term;

        delta_frac = denom_term * num_term;
        cf *= delta_frac;

        coeff = -(alpha + k) * (alpha + beta + k) * x / ((alpha + 2 * k) * (alpha + 2 * k + 1.));
    
        /* second step */
        denom_term = 1. + coeff * denom_term;
        num_term = 1. + coeff / num_term;

        if (fabs(denom_term) < cutoff)
            denom_term = nan("");

        if (fabs(num_term) < cutoff)
            num_term = nan("");

        denom_term = 1. / denom_term;

        delta_frac = denom_term * num_term;
        cf *= delta_frac;

        /* last iteration checks */
        //if (fabs(delta_frac - 1.) < 2. * CUDA_DBL_EPSILON)
        //    break;

        //if (cf * fabs(delta_frac - 1.) < epsabs)
        //    break;
    
    }

    // These checks are originally done within the loop
    // If this logic within the loop is modified, this should be modified as well
    //if (fabs(delta_frac - 1.) < 2. * CUDA_DBL_EPSILON || cf * fabs(delta_frac - 1.) < epsabs)
    //    return nan("");
        
    return cf;

}

__global__ void betacdf_dirCF_kernel(double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double a = alpha;
    double b = beta;
    if (idx < size){
        double my_x = x[idx];

        double ln_beta = lgamma(a + b) - lgamma(a) - lgamma(b);
        double ln_pre = -ln_beta + a * log(my_x) + b * log1p(-my_x);
        double prefactor = exp(ln_pre);

        double epsabs = 0.;
        double cf = cuda_beta_cont_frac(a, b, my_x, epsabs);

        y[idx] = prefactor * cf / a;
        
    }
}

__global__ void betacdf_hypergeoCF_kernel(double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double a = alpha;
    double b = beta;
    if (idx < size){
        double my_x = x[idx];

        double ln_beta = lgamma(a + b) - lgamma(a) - lgamma(b);
        double ln_pre = -ln_beta + a * log(my_x) + b * log1p(-my_x);
        double prefactor = exp(ln_pre);

        double epsabs = 1. / (prefactor / b) * CUDA_DBL_EPSILON;
        double cf = cuda_beta_cont_frac(b, a, 1. - my_x, epsabs);

        double term = prefactor * cf / b;

        y[idx] = 1. - term;
    }
}

// https://github.com/ampl/gsl/blob/master/specfunc/gamma_inc.c#L500
__global__ void betacdf_la_sb_kernel(double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double a = alpha;
    double b = beta;
    if (idx < size){
        double my_x = x[idx];
        double N = a + (b - 1.) / 2.;
        my_x = -N * log1p(-my_x);

        y[idx] = nan("");
        return;
    }
}

// https://github.com/ampl/gsl/blob/master/specfunc/gamma_inc.c#L581
__global__ void betacdf_sa_lb_kernel_f(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float a = alpha;
    float b = beta;
    if (idx < size){
        float my_x = x[idx];
        float N = b + (a - 1.) / 2.;
        my_x = -N * log1pf(-my_x);

        y[idx] = nanf("");
        return;
    }
}



// Define a type for the function that launches the kernel
typedef void (*KernelLauncher)(double*, double*, double, double, int, int);
typedef void (*KernelLauncherFloat)(float*, float*, float, float, int, int);

inline void launch_betapdf_kernel(double *d_x, double *d_y, double alpha, double beta, int size, int block_size) {
    int n_blocks = size / block_size + (size % block_size == 0 ? 0 : 1);
    betapdf_kernel<<<n_blocks, block_size>>>(d_x, d_y, alpha, beta, size);
}

inline void launch_betapdf_kernel_f(float *d_x, float *d_y, float alpha, float beta, int size, int block_size) {
    int n_blocks = size / block_size + (size % block_size == 0 ? 0 : 1);
    betapdf_kernel_f<<<n_blocks, block_size>>>(d_x, d_y, alpha, beta, size);
}

inline void launch_betacdf_kernel(double *d_x, double *d_y, double alpha, double beta, int size, int block_size) {
    int n_blocks = size / block_size + (size % block_size == 0 ? 0 : 1);
    n_blocks *= 1; // Just to suppress tmp warning
    //TODO: Implement smthing with the kernel
}

void beta_array_cuda(const double *x, double *y, const double alpha, const double beta, unsigned long size, KernelLauncher kernel_launcher){
    
    #ifdef DEBUG
    hipEvent_t t1, t2, t3, t4;
    float elapsedMemcpyCG, elapsedKernel, elapsedMemcpyGC, elapsedTotal;
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventCreate(&t3);
    hipEventCreate(&t4);

    hipEventRecord(t1, 0);
    #endif

    // Allocate memory on the device
    double *d_x, *d_y;

    hipMalloc(&d_x, size * sizeof(double));
    hipMalloc(&d_y, size * sizeof(double));

    // Copy the data to the device
    hipMemcpy(d_x, x, size * sizeof(double), hipMemcpyHostToDevice);

    #ifdef DEBUG
    hipEventRecord(t2, 0);
    hipEventSynchronize(t2);
    hipEventElapsedTime(&elapsedMemcpyCG, t1, t2);
    #endif

    // Launch the kernel
    int block_size = 256;
    kernel_launcher(d_x, d_y, alpha, beta, size, block_size);

    #ifdef DEBUG
    hipEventRecord(t3, 0);
    hipEventSynchronize(t3);
    hipEventElapsedTime(&elapsedKernel, t2, t3);
    #endif

    // Copy the result back to the host
    hipMemcpy(y, d_y, size * sizeof(double), hipMemcpyDeviceToHost);

    // Free the memory on the device
    hipFree(d_x);
    hipFree(d_y);

    #ifdef DEBUG
    hipEventRecord(t4, 0);
    hipEventSynchronize(t4);
    hipEventElapsedTime(&elapsedMemcpyGC, t3, t4);
    hipEventElapsedTime(&elapsedTotal, t1, t4);

    cerr << "Full function time(events) = " << elapsedTotal / 1000 << endl;
    cerr << "\tKernel execution time = " << elapsedKernel / 1000 << endl;
    cerr << "\tMemory transfer time = " << elapsedMemcpyCG / 1000 << " + " << elapsedMemcpyGC / 1000 << endl;
    #endif

    return;
}

void beta_array_cuda_float(const float *x, float *y, const float alpha, const float beta, unsigned long size, KernelLauncherFloat kernel_launcher){
    
    #ifdef DEBUG
    hipEvent_t t1, t2, t3, t4;
    float elapsedMemcpyCG, elapsedKernel, elapsedMemcpyGC, elapsedTotal;
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventCreate(&t3);
    hipEventCreate(&t4);

    hipEventRecord(t1, 0);
    #endif

    // Allocate memory on the device
    float *d_x, *d_y;

    hipMalloc(&d_x, size * sizeof(float));
    hipMalloc(&d_y, size * sizeof(float));

    // Copy the data to the device
    hipMemcpy(d_x, x, size * sizeof(float), hipMemcpyHostToDevice);

    #ifdef DEBUG
    hipEventRecord(t2, 0);
    hipEventSynchronize(t2);
    hipEventElapsedTime(&elapsedMemcpyCG, t1, t2);
    #endif

    // Launch the kernel
    int block_size = 256;
    kernel_launcher(d_x, d_y, alpha, beta, size, block_size);

    #ifdef DEBUG
    hipEventRecord(t3, 0);
    hipEventSynchronize(t3);
    hipEventElapsedTime(&elapsedKernel, t2, t3);
    #endif

    // Copy the result back to the host
    hipMemcpy(y, d_y, size * sizeof(float), hipMemcpyDeviceToHost);

    // Free the memory on the device
    hipFree(d_x);
    hipFree(d_y);

    #ifdef DEBUG
    hipEventRecord(t4, 0);
    hipEventSynchronize(t4);
    hipEventElapsedTime(&elapsedMemcpyGC, t3, t4);
    hipEventElapsedTime(&elapsedTotal, t1, t4);

    cerr << "Full function time(events) = " << elapsedTotal / 1000 << endl;
    cerr << "\tKernel execution time = " << elapsedKernel / 1000 << endl;
    cerr << "\tMemory transfer time = " << elapsedMemcpyCG / 1000 << " + " << elapsedMemcpyGC / 1000 << endl;
    #endif

    return;
}

// CUDA kernel launch to compute the beta distribution
void betapdf_cuda(const double *x, double *y, const double alpha, const double beta, unsigned long size){
    
    beta_array_cuda(x, y, alpha, beta, size, launch_betapdf_kernel);

    return;
}

// CUDA kernel launch to compute the beta distribution
void betapdf_cuda(const float *x, float *y, const float alpha, const float beta, unsigned long size){

    beta_array_cuda_float(x, y, alpha, beta, size, launch_betapdf_kernel_f);

    return;
}

void betacdf_cuda(const double *x, double *y, const double alpha, const double beta, unsigned long size){
    
    beta_array_cuda(x, y, alpha, beta, size, launch_betacdf_kernel);

    return;
}
