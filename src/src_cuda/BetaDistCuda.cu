#include "hip/hip_runtime.h"
#include "BetaDistCuda.hpp"

#include <hip/hip_fp16.h>

#include <chrono>
#include <iostream>

using std::cerr;
using std::endl;

using profile_clock_t = std::chrono::high_resolution_clock;
using profile_duration_t = std::chrono::duration<double>;

__global__ void betapdf_kernel(double *x, double *y, double alpha, double beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = pow(x[idx], alpha - 1) * pow(1 - x[idx], beta - 1) * exp(lgamma(alpha + beta) - lgamma(alpha) - lgamma(beta));
    }
}

__global__ void betapdf_kernel_f(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}

__global__ void betapdf_kernel_h(float *x, float *y, float alpha, float beta, size_t size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        y[idx] = powf(x[idx], alpha - 1) * powf(1 - x[idx], beta - 1) * expf(lgammaf(alpha + beta) - lgammaf(alpha) - lgammaf(beta));
    }
}

// CUDA kernel launch to compute the beta distribution
std::vector<double> betapdf_cuda(std::vector<double> x, double alpha, double beta, GPU_Type precision){
    // Allocate memory on the device
    double *d_x, *d_y;
    float *d_x_f, *d_y_f, alpha_f, beta_f;
    if (precision == GPU_Type::DOUBLE){
        hipMalloc(&d_x, x.size() * sizeof(double));
        hipMalloc(&d_y, x.size() * sizeof(double));
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        alpha_f = (float)alpha;
        beta_f = (float)beta;
        hipMalloc(&d_x_f, x.size() * sizeof(float));
        hipMalloc(&d_y_f, x.size() * sizeof(float));
    }

    // Copy the data to the device
    if (precision == GPU_Type::DOUBLE){
        hipMemcpy(d_x, x.data(), x.size() * sizeof(double), hipMemcpyHostToDevice);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> x_f(x.begin(), x.end());
        hipMemcpy(d_x_f, x_f.data(), x_f.size() * sizeof(float), hipMemcpyHostToDevice);
    }

    // Launch the kernel
    int block_size = 256;
    int n_blocks = x.size() / block_size + (x.size() % block_size == 0 ? 0 : 1);
    if (precision == GPU_Type::DOUBLE)
        betapdf_kernel<<<n_blocks, block_size>>>(d_x, d_y, alpha, beta, x.size());
    if (precision == GPU_Type::FLOAT)
        betapdf_kernel_f<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha, beta, x.size());
    if (precision == GPU_Type::HALF)
        betapdf_kernel_h<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha_f, beta_f, x.size());

    // Copy the result back to the host
    std::vector<double> y(x.size());
    if (precision == GPU_Type::DOUBLE)
        hipMemcpy(y.data(), d_y, x.size() * sizeof(double), hipMemcpyDeviceToHost);
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> y_f(x.size());
        hipMemcpy(y_f.data(), d_y_f, x.size() * sizeof(float), hipMemcpyDeviceToHost);
        y = std::vector<double>(y_f.begin(), y_f.end());
    }

    // Free the memory on the device
    if (precision == GPU_Type::DOUBLE){
        hipFree(d_x);
        hipFree(d_y);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        hipFree(d_x_f);
        hipFree(d_y_f);
    }

    return y;
}

std::vector<double> betapdf_cuda_times(std::vector<double> x, double alpha, double beta, GPU_Type precision){
    auto t1 = profile_clock_t::now();
    // Allocate memory on the device
    double *d_x, *d_y;
    float *d_x_f, *d_y_f, alpha_f, beta_f;
    if (precision == GPU_Type::DOUBLE){
        hipMalloc(&d_x, x.size() * sizeof(double));
        hipMalloc(&d_y, x.size() * sizeof(double));
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        alpha_f = (float)alpha;
        beta_f = (float)beta;
        hipMalloc(&d_x_f, x.size() * sizeof(float));
        hipMalloc(&d_y_f, x.size() * sizeof(float));
    }

    // Copy the data to the device
    if (precision == GPU_Type::DOUBLE){
        hipMemcpy(d_x, x.data(), x.size() * sizeof(double), hipMemcpyHostToDevice);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> x_f(x.begin(), x.end());
        hipMemcpy(d_x_f, x_f.data(), x_f.size() * sizeof(float), hipMemcpyHostToDevice);
    }

    auto t2 = profile_clock_t::now();

    // Launch the kernel
    int block_size = 256;
    int n_blocks = x.size() / block_size + (x.size() % block_size == 0 ? 0 : 1);
    if (precision == GPU_Type::DOUBLE)
        betapdf_kernel<<<n_blocks, block_size>>>(d_x, d_y, alpha, beta, x.size());
    if (precision == GPU_Type::FLOAT)
        betapdf_kernel_f<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha, beta, x.size());
    if (precision == GPU_Type::HALF)
        betapdf_kernel_h<<<n_blocks, block_size>>>(d_x_f, d_y_f, alpha_f, beta_f, x.size());

    auto t3 = profile_clock_t::now();

    // Copy the result back to the host
    std::vector<double> y(x.size());
    if (precision == GPU_Type::DOUBLE)
        hipMemcpy(y.data(), d_y, x.size() * sizeof(double), hipMemcpyDeviceToHost);
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        std::vector<float> y_f(x.size());
        hipMemcpy(y_f.data(), d_y_f, x.size() * sizeof(float), hipMemcpyDeviceToHost);
        y = std::vector<double>(y_f.begin(), y_f.end());
    }

    // Free the memory on the device
    if (precision == GPU_Type::DOUBLE){
        hipFree(d_x);
        hipFree(d_y);
    }
    if (precision == GPU_Type::FLOAT || precision == GPU_Type::HALF){
        hipFree(d_x_f);
        hipFree(d_y_f);
    }

    auto t4 = profile_clock_t::now();

    cerr << "Full function time = " << profile_duration_t(t4 - t1).count() << endl;
    cerr << " Kernel execution time = " << profile_duration_t(t3 - t2).count() << endl;
    cerr << " Memory transfer time = " << profile_duration_t(t2 - t1).count() << " + " << profile_duration_t(t4 - t3).count() << endl;

    return y;
}

std::vector<double> betacdf_cuda(std::vector<double> x, double alpha, double beta){
    return std::vector<double>();
}
